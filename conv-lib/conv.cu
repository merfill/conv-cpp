#include "hip/hip_runtime.h"

#include "blur_impl.h"

#include <hip/hip_runtime.h>

#include <cstdint>
#include <vector>
#include <stdexcept>
#include <memory>
#include <cmath>
#include <iostream>


__global__
void gaussian_blur_rgb(const uint8_t* in_img, uint8_t* out_img, int height, int width, int ksize, const float* kernel) {
  // Current pixel coordinates
  int r = blockIdx.x * blockDim.x + threadIdx.x;
  int c = blockIdx.y * blockDim.y + threadIdx.y;

  RGB_BLUR();
}

__global__
void gaussian_blur_grey(const uint8_t* in_img, uint8_t* out_img, int height, int width, int ksize, const float* kernel) {
  // Current pixel coordinates
  int c =  blockIdx.y * blockDim.y + threadIdx.y;
  int r =  blockIdx.x * blockDim.x + threadIdx.x;

  GREY_BLUR();
}

#define CUDA_CHECK(call) {\
  {\
    hipError_t err = call;\
    if (hipSuccess != err) {\
      throw std::runtime_error(hipGetErrorString(err));\
    }\
  }\
}

namespace conv {

std::vector<float> get_gaussian_kernel(float sigma, size_t ksize) {
  // Variance inherited from sigma value
  float variance = 2. * sigma * sigma;

  // Sum of all kernel values for the normalization
  float sum = 0.;

  // Generate pure kernel values
  std::vector<float> kernel(ksize * ksize, 0.);
  size_t kindex = 0; // kernel index
  int mean = ksize / 2;
  for (int x = 0; x < ksize; ++x) {
    for (int y = 0; y < ksize; ++y) {
      float xr = x - mean, yr = y - mean;
      float r = sqrtf(xr * xr + yr * yr);
      kernel[kindex] = (std::exp(-(r * r) / variance) )/ (3.141592 * variance);
      sum += kernel[kindex];
      ++kindex;
    }
  }

  // Normalize kernel values
  if (sum < 0.e-6) {
    sum = 0.e-6;
  }
  float s = 0.;
  for (auto i = 0; i < ksize * ksize; ++i) {
    kernel[i] /= sum;
    s += kernel[i];
  }

  return kernel;
}

int ceiling(const int all_size, const int block_size) {
  return (all_size + block_size - 1) / block_size;
}

void gaussian_blur_gpu(const uint8_t* in_img, uint8_t* out_img, size_t height, size_t width, size_t depth, float sigma, size_t ksize) {
  // Device memory
  uint8_t *d_src = 0, *d_dst = 0;
  float *d_kernel = 0;
  size_t bytes = height * width * depth;

  typedef std::unique_ptr<uint8_t, hipError_t(*)(void*)> cuda_uint8_ptr;
  typedef std::unique_ptr<float, hipError_t(*)(void*)> cuda_float_ptr;

  // Allocate memory on GPU
  CUDA_CHECK(hipMalloc(&d_src, bytes));
  cuda_uint8_ptr d_src_ptr(d_src, hipFree);
  CUDA_CHECK(hipMalloc(&d_dst, bytes));
  cuda_uint8_ptr d_dst_ptr(d_dst, hipFree);
  CUDA_CHECK(hipMalloc(&d_kernel, ksize * ksize * sizeof(float)));
  cuda_float_ptr d_kernel_ptr(d_kernel, hipFree);

  // Generate kernel
  auto kernel = get_gaussian_kernel(sigma, ksize);

  // Copy host data to device memory
  CUDA_CHECK(hipMemcpy(d_src, in_img, bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_dst, 0, bytes));
  CUDA_CHECK(hipMemcpy(d_kernel, (void*)&kernel[0], kernel.size() * sizeof(float), hipMemcpyHostToDevice));

  // Calculate convolution
  const int block_size = 4;
  int grid_x = ceiling(height, block_size);
  int grid_y = ceiling(width, block_size);
  dim3 block(block_size, block_size);
  dim3 grid(grid_x, grid_y);
  switch (depth) {
    case 1:
      gaussian_blur_grey<<<grid, block>>>(d_src, d_dst, height, width, ksize, d_kernel);
      break;
    case 3:
      gaussian_blur_rgb<<<grid, block>>>(d_src, d_dst, height, width, ksize, d_kernel);
      break;
    default:
      throw std::runtime_error("incorrect color depth passed. Must be 1 or 3");
  }

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy data back to host
  CUDA_CHECK(hipMemcpy(&out_img[0], d_dst, bytes, hipMemcpyDeviceToHost));
}

void gaussian_blur_cpu(const uint8_t* in_img, uint8_t* out_img, size_t height, size_t width, size_t depth, float sigma, size_t ksize) {
  auto kernel = get_gaussian_kernel(sigma, ksize);
  for (int r = 0; r < height; ++r) {
    for (int c = 0; c < width; ++c) {
      RGB_BLUR();
    }
  }
}

} // namespace conv

